#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>

#include "util.cuh"
#include "operator.cuh"
#include "spmm.h"

// Memory & time efficient implementation of generalized spmm
// Much of the code is inspired by GE-SpMM
// https://github.com/hgyhungry/ge-spmm

namespace at {

namespace {

const int kCoarseningFactor = 2;
const int kThreadPerBlock = 256;

} // namespace anonymous

template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void spmm_forward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const scalar_t *value,
                           const scalar_t *input, scalar_t *output,
                           int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    scalar_t *value_buf = reinterpret_cast<scalar_t *>(col_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    value_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;
    scalar_t out[kCoarseningFactor];
#pragma unroll
    for (int64_t i = 0; i < kCoarseningFactor; i++)
        out[i] = NaryOp::zero;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            value_buf[threadIdx.x] = value[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            scalar_t val = value_buf[offset_ptr];
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t x = BinaryOp::forward(val, input[col * dim + d]);
                out[i] = NaryOp::forward(out[i], x);
            }
        }
        __syncwarp();
    }

#pragma unroll
    for (int64_t i = 0; i < kCoarseningFactor; i++) {
        int64_t d = d_start + i * warpSize;
        if (d >= dim)
            break;
        output[row * dim + d] = out[i];
    }
}

// both sparse and input require gradients
template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void spmm_backward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const scalar_t *value,
                            const scalar_t *input, const scalar_t *output, const scalar_t *output_grad,
                            scalar_t *value_grad, scalar_t *input_grad,
                            int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    scalar_t *value_buf = reinterpret_cast<scalar_t *>(col_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    value_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            value_buf[threadIdx.x] = value[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            scalar_t val = value_buf[offset_ptr];
            scalar_t val_grad = 0;
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t in = input[col * dim + d];
                scalar_t out = output[row * dim + d];
                scalar_t out_grad = output_grad[row * dim + d];
                scalar_t x = BinaryOp::forward(val, in);
                scalar_t dx_dval = BinaryOp::backward_lhs(val, in);
                scalar_t dx_din = BinaryOp::backward_rhs(val, in);
                scalar_t dout_dx = NaryOp::backward(out, x);
                val_grad += out_grad * dout_dx * dx_dval;
                atomicAdd(&input_grad[col * dim + d], out_grad * dout_dx * dx_din);
            }
            val_grad = warp_reduce(val_grad);
            if (threadIdx.x == 0)
                atomicAdd(&value_grad[block_ptr + offset_ptr], val_grad);
        }
        __syncwarp();
    }
}

// only input requires gradients
template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void spmm_backward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const scalar_t *value,
                            const scalar_t *input, const scalar_t *output, const scalar_t *output_grad,
                            scalar_t *input_grad,
                            int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    scalar_t *value_buf = reinterpret_cast<scalar_t *>(col_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    value_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            value_buf[threadIdx.x] = value[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            scalar_t val = value_buf[offset_ptr];
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t in = input[col * dim + d];
                scalar_t out = output[row * dim + d];
                scalar_t out_grad = output_grad[row * dim + d];
                scalar_t x = BinaryOp::forward(val, in);
                scalar_t dx_din = BinaryOp::backward_rhs(val, in);
                scalar_t dout_dx = NaryOp::backward(out, x);
                atomicAdd(&input_grad[col * dim + d], out_grad * dout_dx * dx_din);
            }
        }
        __syncwarp();
    }
}

template <template<class> class NaryOp, template<class> class BinaryOp>
Tensor spmm_forward_cuda(const SparseTensor &sparse, const Tensor &input_) {
    constexpr const char *fn_name = "spmm_forward_cuda";
    TensorArg sparse_arg(sparse, "sparse", 1), input_arg(input_, "input", 2);

    spmm_forward_check(fn_name, sparse_arg, input_arg);
    checkAllSameGPU(fn_name, {sparse_arg, input_arg});

    const Tensor input = input_.contiguous();

    int64_t nnz = sparse._nnz();
    int64_t dim = input.size(1);
    int64_t num_row = sparse.size(0);
    Tensor output = at::empty({num_row, dim}, input.options());

    auto csr = coo2csr(sparse);
    Tensor row_ptr = std::get<0>(csr);
    Tensor col_ind = std::get<1>(csr);
    Tensor value = std::get<2>(csr);

    hipSetDevice(input.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const int dim_per_block = 32; // warpSize
    const int num_dim_block = (dim + dim_per_block * kCoarseningFactor - 1) / (dim_per_block * kCoarseningFactor);
    const int row_per_block = kThreadPerBlock / dim_per_block;
    const int num_row_block = (num_row + row_per_block - 1) / row_per_block;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "spmm_forward_cuda", [&] {
        const int memory_size = kThreadPerBlock * (sizeof(int64_t) + sizeof(scalar_t));
        spmm_forward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
            <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
            row_ptr.data_ptr<int64_t>(),
            col_ind.data_ptr<int64_t>(),
            value.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            num_row, nnz, dim
        );
    });

    return output;
}

template <template<class> class NaryOp, template<class> class BinaryOp>
std::tuple<SparseTensor, Tensor> spmm_backward_cuda(
        const SparseTensor &sparse, const Tensor &input_, const Tensor &output_, const Tensor &output_grad_) {
    constexpr const char *fn_name = "spmm_backward_cuda";
    TensorArg sparse_arg(sparse, "sparse", 1), input_arg(input_, "input", 2), output_arg(output_, "output", 3),
              output_grad_arg(output_grad_, "output_grad", 4);

    spmm_backward_check(fn_name, sparse_arg, input_arg, output_arg, output_grad_arg);
    checkAllSameGPU(fn_name, {sparse_arg, input_arg, output_arg, output_grad_arg});

    const Tensor input = input_.contiguous();
    const Tensor output = output_.contiguous();
    const Tensor output_grad = output_grad_.contiguous();

    int64_t nnz = sparse._nnz();
    int64_t dim = input.size(1);
    int64_t num_row = sparse.size(0);
    Tensor value_grad = at::zeros_like(sparse.values());
    Tensor input_grad = at::zeros_like(input);
    SparseTensor sparse_grad = at::_sparse_coo_tensor_unsafe(sparse.indices(), value_grad, sparse.sizes());

    auto csr = coo2csr(sparse);
    Tensor row_ptr = std::get<0>(csr).contiguous();
    Tensor col_ind = std::get<1>(csr).contiguous();
    Tensor value = std::get<2>(csr).contiguous();

    hipSetDevice(input.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const int dim_per_block = 32; // warpSize
    const int num_dim_block = (dim + dim_per_block * kCoarseningFactor - 1) / (dim_per_block * kCoarseningFactor);
    const int row_per_block = kThreadPerBlock / dim_per_block;
    const int num_row_block = (num_row + row_per_block - 1) / row_per_block;

    if (sparse.requires_grad())
        AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "spmm_backward_cuda", [&] {
            const int memory_size = kThreadPerBlock * (sizeof(int64_t) + sizeof(scalar_t));
            spmm_backward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
                <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
                row_ptr.data_ptr<int64_t>(),
                col_ind.data_ptr<int64_t>(),
                value.data_ptr<scalar_t>(),
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                output_grad.data_ptr<scalar_t>(),
                value_grad.data_ptr<scalar_t>(),
                input_grad.data_ptr<scalar_t>(),
                num_row, nnz, dim
            );
        });
    else
        AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "spmm_backward_cuda", [&] {
            const int memory_size = kThreadPerBlock * (sizeof(int64_t) + sizeof(scalar_t));
            spmm_backward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
                <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
                row_ptr.data_ptr<int64_t>(),
                col_ind.data_ptr<int64_t>(),
                value.data_ptr<scalar_t>(),
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                output_grad.data_ptr<scalar_t>(),
                input_grad.data_ptr<scalar_t>(),
                num_row, nnz, dim
            );
        });

    return std::make_tuple(sparse_grad, input_grad);
}

#define DECLARE_FORWARD_IMPL(ADD, MUL, NARYOP, BINARYOP) \
    Tensor spmm_##ADD##_##MUL##_forward_cuda(const SparseTensor &sparse, const Tensor &input) { \
        return spmm_forward_cuda<NARYOP, BINARYOP>(sparse, input);                              \
    }

#define DECLARE_BACKWARD_IMPL(ADD, MUL, NARYOP, BINARYOP) \
    std::tuple<SparseTensor, Tensor> spmm_##ADD##_##MUL##_backward_cuda(                                        \
            const SparseTensor &sparse, const Tensor &input, const Tensor &output, const Tensor &output_grad) { \
        return spmm_backward_cuda<NARYOP, BINARYOP>(sparse, input, output, output_grad);                        \
    }

DECLARE_FORWARD_IMPL(add, mul, NaryAdd, BinaryMul)
DECLARE_BACKWARD_IMPL(add, mul, NaryAdd, BinaryMul)

DECLARE_FORWARD_IMPL(min, mul, NaryMin, BinaryMul)
DECLARE_BACKWARD_IMPL(min, mul, NaryMin, BinaryMul)

DECLARE_FORWARD_IMPL(max, mul, NaryMax, BinaryMul)
DECLARE_BACKWARD_IMPL(max, mul, NaryMax, BinaryMul)

DECLARE_FORWARD_IMPL(add, add, NaryAdd, BinaryAdd)
DECLARE_BACKWARD_IMPL(add, add, NaryAdd, BinaryAdd)

DECLARE_FORWARD_IMPL(min, add, NaryMin, BinaryAdd)
DECLARE_BACKWARD_IMPL(min, add, NaryMin, BinaryAdd)

DECLARE_FORWARD_IMPL(max, add, NaryMax, BinaryAdd)
DECLARE_BACKWARD_IMPL(max, add, NaryMax, BinaryAdd)

} // namespace at